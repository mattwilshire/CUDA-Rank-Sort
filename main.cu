#include "hip/hip_runtime.h"

// GPU Rank Sort
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <cstdlib>
#include <time.h>
using namespace std;

__host__ void sortOnHost(int n, int* h_a, int* h_b) {

    for (int i = 0; i < n; i++) {
        int rank = 0;
        for (int j = 0; j < n; j++) {
            if (h_a[i] > h_a[j])
                rank++;
        }
        h_b[rank] = h_a[i];
    }
}

__global__ void sortOnDevice1(int n, int* d_a, int* d_b) {
    // Rank sort but we access global memory e.g d_a[threadId], d_a[j]
    // iteration i computed by threadId
    int threadId = threadIdx.x + blockIdx.x * blockDim.x;

    if (threadId < n) {
        int rank = 0;
        for (int j = 0; j < n; j++) {
            // d_a[threadId] accessed, slow.
            if (d_a[threadId] > d_a[j])
                rank++;
        }
        // d_a[threadId] accessed again, slow!!!
        d_b[rank] = d_a[threadId];

    }
}

__global__ void sortOnDevice2(int n, int* d_a, int* d_b) {
    // Again but only access global memory once
    int threadId = threadIdx.x + blockIdx.x * blockDim.x;

    if (threadId < n) {
        int rank = 0, elem = d_a[threadId]; // Accessed once 
        for (int j = 0; j < n; j++) {
            if (elem > d_a[j]) // Yes, we are accessing it here but we need to for comparison
                rank++;
        }
        d_b[rank] = elem;

    }
}

__global__ void sortOnDevice3(int n, int* d_a, int* d_b) {
    // Fastest approach using shared memory in the GPU
    int threadId = threadIdx.x + blockIdx.x * blockDim.x;

    // Shared memory, scoped to the block of threads
    extern __shared__ int share_a[];

    // copy d_a to share_a
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        share_a[i] = d_a[i];
    }

    // Wait for shared memory to be filled up
    __syncthreads();

    if (threadId < n) {
        int rank = 0, elem = share_a[threadId]; // Pull from fast shared memory :)
        for (int j = 0; j < n; j++) {
            if (elem > share_a[j]) // Comparison using fast shared memory
                rank++;
        }
        d_b[rank] = elem;

    }
}


int main(int argc, char** argv) {
    // n number of elements in the array
    int n = 100000, blockSize = 512;

    // allocate the arrays and initialise h_a
    int* h_a = (int*)malloc(n * sizeof(int));
    int* h_b = (int*)malloc(n * sizeof(int));

    int* d_a, * d_b;
    hipMalloc((void**)&d_a, n * sizeof(int));
    hipMalloc((void**)&d_b, n * sizeof(int));

    // Fill in reverse to ensure nothing is sorted.
    for (int i = 0; i < n; i++)
        h_a[i] = n - i;

    clock_t time1, time2;

    // copy h_a to d_a
    hipMemcpy(d_a, h_a, n * sizeof(int), hipMemcpyHostToDevice);

    /*
        HOST SORT
    */

    time1 = clock();
    sortOnHost(n, h_a, h_b);
    time2 = clock();
    printf("Host Sort: %lf\n", 1.0 * (time2 - time1) / CLOCKS_PER_SEC);

    /*
        GPU SORT
    */
    int gridSize = (n + blockSize - 1) / blockSize;
    time1 = clock();
    sortOnDevice1 << <gridSize, blockSize >> > (n, d_a, d_b);
    hipDeviceSynchronize(); // Wait for GPU to finish
    time2 = clock();
    printf("GPU Sort using Global Memory: %lf\n", 1.0 * (time2 - time1) / CLOCKS_PER_SEC);

    time1 = clock();
    sortOnDevice2 << <gridSize, blockSize >> > (n, d_a, d_b);
    hipDeviceSynchronize();
    time2 = clock();
    printf("GPU Sort storing global memory value in local variable: %lf\n", 1.0 * (time2 - time1) / CLOCKS_PER_SEC);
    
    time1 = clock();
    sortOnDevice3 << <gridSize, blockSize >> > (n, d_a, d_b);
    hipDeviceSynchronize(); 
    time2 = clock();
    printf("GPU Sort Shared Memory: %lf\n", 1.0 * (time2 - time1) / CLOCKS_PER_SEC);

    // copy d_b to h_b
    hipMemcpy(h_b, d_b, n * sizeof(int), hipMemcpyDeviceToHost);

    // Print out first 100 elements to make sure they are sorted
    //for(int i = 0; i < 100; i++)
        //printf("%d ", h_b[i]);

    // No memory leaks this time around, i want my games to still have some VRAM...
    hipFree(d_a);
    hipFree(d_b);

    scanf("%d");
    return 0;
}